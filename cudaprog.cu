#include "hip/hip_runtime.h"
#define P 256

__global__ void kernel(float *a, float *b, int c){

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ s_A;
	
	if(tid < n){
		
		if(P*P < n){
			int pom = (n / P*P);
			for(int i = 0; i < pom; i++){
				s_A[tid*pom + i] = a[tid*pom + i]; 
				__syncthreads();
			}
			int pom2 = n % P*P;
			if(tid < pom2){
				s_A[P*P*pom + tid] = a[P*P*pom + tid];
				__syncthreads();
			}
			
			
			if(tid < (n-2)){
				if(P*P >= (n-2)){
					b[tid] = (2*s_A[tid] + 3*s_A[tid+1] + 4*s_A[tid+2])/9.f;
				}
				else{
					int pom3 = (n-2) / P*P;
					int pom4 = (n-2) % P*P;
					for(int i = 0; i < pom3; i++){
						b[tid*pom3 + i] = (2*s_A[tid*pom3 + i] + 3*s_A[tid*pom3 + i +1] + 4*s_A[tid*pom3 + i +2])/9.f;
					}
					if(tid < pom4){
						b[P*P*pom3 + tid] = (2*s_A[P*P*pom3 + tid] + 3*s_A[P*P*pom3 + tid + 1] + 4*s_A[P*P*pom3 + tid +2])/9.f;
					}
				}
			}
		}
		else{
			s_A[tid] = a[tid];
			__syncthreads();
			
			if(tid < n-2){
				b[tid] = (2*s_A[tid] + 3*s_A[tid+1] + 4*s_A[tid+2])/9.f;
			}
		}
	}
}

__host__ int main(){

	int n;
	float *h_A, *h_B;
	float *d_A, *d_B;
	
	scanf("%d", &n);
	h_A = (int*)malloc(n * sizeof(float));
	h_B = (int*)malloc(n * sizeof(float));
	
	for(int i = 0; i < n; i++){
		h_A[i] = rand() % 10;
	}
	
	hipMalloc((void**)&d_A, n * sizeof(float));
	hipMalloc((void**)&d_B, n * sizeof(float));
	
	hipMemcpy(d_A, h_A, n * sizeof(float), hipMemcpyHostToDevice);
	
	kernel<<< P, P >>>(d_A,d_B,n);
	
	hipMemcpy(h_B, d_B, n * sizeof(float), hipMemcpyDeviceToHost);
	
	free(h_A);
	free(h_B);
	hipFree(d_A);
	hipFree(d_B);
	return 0;
{
